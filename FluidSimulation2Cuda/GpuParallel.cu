#include "hip/hip_runtime.h"
#include "GpuParallel.cuh"
#include "CudaMath.cuh"

#include <chrono>
#include "Phisics.hpp"

#include <hip/hip_runtime.h>

constexpr float HOW_FAR_INTO_THE_FUTURE = 10.0f;

constexpr int maxThreadsPerBlock = 512;

struct Range {
	int start;
	int end;
};


Particle* deviceParticles;

Range* lengths;

Surface2D* deviceObstacles;

Particle* deviceInteractionMatrixParticles;

int* deviceLengths;
int interactionMatrixSize;

size_t maxParticlesInInteractionMatrixCell;
size_t interactionMatrixRows;
size_t interactionMatrixCols;

__device__ int counterDensitiesDone = 0;
__device__ int counterPredictedPositionsDone = 0;
__device__ int counterFutureVelocitiesDone = 0;


__device__ Range getParticlesInCell(Vector2D position, int particleRadiusOfRepel,
	int* lengths, size_t interactionMatrixRows, size_t interactionMatrixCols,
	size_t maxParticlesInInteractionMatrixCell)
{
	int row = position.Y / particleRadiusOfRepel;
	int col = position.X / particleRadiusOfRepel;

	if (row < 0 || row >= interactionMatrixRows || col < 0 || col >= interactionMatrixCols) {
		return;
	}

	int start = (row * interactionMatrixCols + col) * maxParticlesInInteractionMatrixCell;
	int end = start + lengths[row * interactionMatrixCols + col];

	return Range{ start, end };
}

__global__ void updateParticleDensitiesKernel(Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	Particle* interactionMatrixParticles, int* lengths, size_t interactionMatrixRows, size_t interactionMatrixCols,
	size_t maxParticlesInInteractionMatrixCell) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= praticlesSize) {
		return;
	}

	//printf("index: %d \n", index);

	Particle particle = particles[index];

	Vector2D point = particle.m_PredictedPosition;

	constexpr auto scalar = 1000;

	float density = 0.0f;
	const float mass = 1.0f;

	Range range = getParticlesInCell(point, particleRadiusOfRepel, lengths, interactionMatrixRows, interactionMatrixCols, maxParticlesInInteractionMatrixCell);

	//printf("index: %d, range.start: %d, range.end: %d \n", index, range.start, range.end);

	for (int i = range.start; i < range.end; i++) {
		Particle otherParticle = interactionMatrixParticles[i];
		float distance = sqrt(CudaMath::squared_distance(point, otherParticle.m_PredictedPosition));
		float influence = CudaMath::smoothingKernel(particleRadiusOfRepel, distance);
		density += mass * influence;
	}

	float volume = 3.1415f * pow(particleRadiusOfRepel, 2);

	density = density / volume * scalar;

	particles[index].m_Density = density;
}

__global__ void updateParticleDensitiesKernel2(Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	Particle* interactionMatrixParticles, int* lengths, size_t interactionMatrixRows, size_t interactionMatrixCols,
	size_t maxParticlesInInteractionMatrixCell) {

	//int index = blockIdx.x * blockDim.x + threadIdx.x;

	int index = blockIdx.x;

	int otherParticleIndex = threadIdx.x;

	if (index >= praticlesSize) {
		return;
	}

	//printf("index: %d \n", index);

	Particle particle = particles[index];

	Vector2D point = particle.m_PredictedPosition;

	constexpr auto scalar = 1000;

	//float density = 0.0f;
	const float mass = 1.0f;

	Range range = getParticlesInCell(point, particleRadiusOfRepel, lengths, interactionMatrixRows, interactionMatrixCols, maxParticlesInInteractionMatrixCell);

	//printf("index: %d, range.start: %d, range.end: %d \n", index, range.start, range.end);

	int i = range.start + otherParticleIndex;

	if (i >= range.end) {
		return;
	}

	__shared__ float sharedDensity[1024];

	Particle otherParticle = interactionMatrixParticles[i];
	float distance = sqrt(CudaMath::squared_distance(point, otherParticle.m_PredictedPosition));
	float influence = CudaMath::smoothingKernel(particleRadiusOfRepel, distance);

	float localDensity = mass * influence;

	//particles[index].m_Density += localDensity;
	atomicAdd(&particles[index].m_Density, localDensity);

	// Synchronize all threads in the block
	__syncthreads();

	if (otherParticleIndex != 0) {
		return;
	}

	float volume = 3.1415f * pow(particleRadiusOfRepel, 2);

	particles[index].m_Density = particles[index].m_Density / volume * scalar;

	//printf("index: %d, density: %f \n", index, particles[index].m_Density);
}

__device__ void updateParticle(int index, Particle* particle, double dt) {
	if (dt == 0) {
		return;
	}

	particle[index].m_LastSafePosition = particle[index].m_Position;

	GpuVector2D newVelocity{ 0,0 };

	newVelocity = GpuVector2D(particle[index].m_Velocity) + GpuVector2D(0.0f, GRAVITY) * dt;

	newVelocity += GpuVector2D(particle[index].m_TemporaryVelocity);

	particle[index].m_Velocity.X = newVelocity.X;
	particle[index].m_Velocity.Y = newVelocity.Y;

	particle[index].m_TemporaryVelocity.X = 0;
	particle[index].m_TemporaryVelocity.Y = 0;

	particle[index].m_Position.X += particle[index].m_Velocity.X * dt;
	particle[index].m_Position.Y += particle[index].m_Velocity.Y * dt;
}

__device__ GpuVector2D calculatePressureForce(Particle particle, int particleRadiusOfRepel, int particleRadius,
	Particle* interactionMatrixParticles, int* lengths, int interactionMatrixRows, int interactionMatrixCols,
	int maxParticlesInInteractionMatrixCell)
{

	GpuVector2D pressureForce = GpuVector2D();
	const float mass = 1.0f;

	Range range = getParticlesInCell(particle.m_PredictedPosition, particleRadiusOfRepel, lengths, interactionMatrixRows, interactionMatrixCols, maxParticlesInInteractionMatrixCell);

	for (int i = range.start; i < range.end; i++) {
		Particle otherParticle = interactionMatrixParticles[i];

		if (particle.m_ID == otherParticle.m_ID) {
			continue;
		}

		float distance = sqrt(CudaMath::squared_distance(particle.m_PredictedPosition, otherParticle.m_PredictedPosition));

		GpuVector2D dir = distance < particleRadius ? GpuVector2D::getRandomDirection() : (GpuVector2D(otherParticle.m_PredictedPosition) - GpuVector2D(particle.m_PredictedPosition)) / distance;

		float slope = CudaMath::smoothingKernelDerivative(particleRadiusOfRepel, distance);

		float density = otherParticle.m_Density;

		float sharedPressure = CudaMath::calculateSharedPressure(density, otherParticle.m_Density);

		pressureForce += -sharedPressure * dir * slope * mass / density;
	}
	//printf("pressureForceX: %f , pressureForceY: %f\n", pressureForce.X, pressureForce.Y);
	return pressureForce;
}


__global__ void calculateParticleFutureVelocitiesKernel(Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	int particleRadius, Particle* interactionMatrixParticles, int* lengths, size_t interactionMatrixRows,
	size_t interactionMatrixCols, size_t maxParticlesInInteractionMatrixCell, double dt)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= praticlesSize) {
		return;
	}

	//printf("index: %d \n", interactionMatrixCols);

	Particle particle = particles[index];

	if (particle.m_Density == 0) {
		return;
	}

	GpuVector2D pressureForce = calculatePressureForce(particle, particleRadiusOfRepel, particleRadius,
		interactionMatrixParticles, lengths, interactionMatrixRows, interactionMatrixCols,
		maxParticlesInInteractionMatrixCell);

	//pressureForce = GpuVector2D(300, 300);
	GpuVector2D pressureAcceleration = pressureForce / particle.m_Density;

	//Vector2D viscosityForce = calculateViscosityForce(particle);
	GpuVector2D viscosityForce = GpuVector2D();

	GpuVector2D futureVelocity = GpuVector2D(particle.m_Velocity) + pressureAcceleration * dt + viscosityForce * dt;

	//printf("index: %d, futureVelocity: %f %f \n", index, futureVelocity.X, futureVelocity.Y);

	particles[index].m_FutureVelocity.X = futureVelocity.X;
	particles[index].m_FutureVelocity.Y = futureVelocity.Y;


	particles[index].m_Velocity = particles[index].m_FutureVelocity;
	updateParticle(index, particles, dt);
	particles[index].m_PredictedPosition = particles[index].m_Position;
}

__global__ void checkCollisionsKernel(Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	int particleRadius, float particleRepulsionForce, Particle* interactionMatrixParticles, int* lengths,
	int interactionMatrixRows, int interactionMatrixCols, int maxParticlesInInteractionMatrixCell,
	Surface2D* obstacles, int obstaclesSize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= praticlesSize) {
		return;
	}

	//printf("index: %d \n", interactionMatrixCols);

	Particle particle = particles[index];

	for (int i = 0; i < obstaclesSize; i++) {

		Surface2D obstacle = obstacles[i];

		if (CudaMath::check_line_segment_circle_intersection(obstacle.Point1, obstacle.Point2,
			particle.m_Position, particleRadius)) {

			//particle->m_Velocity = reflectionVector * 0.1f;
			particles[index].m_Velocity.X = 0;
			particles[index].m_Velocity.Y = 0;

			/*particle.m_Velocity.Y = -particle.m_Velocity.Y;*/

			particles[index].m_Position = particle.m_LastSafePosition;

			break;
		}
	}

	Range range = getParticlesInCell(particle.m_PredictedPosition, particleRadiusOfRepel, lengths, interactionMatrixRows, interactionMatrixCols, maxParticlesInInteractionMatrixCell);

	//for (auto& otherParticle : m_Particles) {
	for (int i = range.start; i < range.end; i++) {

		Particle otherParticle = interactionMatrixParticles[i];

		if (particle.m_ID == otherParticle.m_ID) {
			continue;
		}

		if (CudaMath::squared_distance(particle.m_Position, otherParticle.m_Position) <= (particleRadius * particleRadius) * 4) {

			GpuVector2D normalVector{};
			normalVector.X = otherParticle.m_Position.X - particle.m_Position.X;
			normalVector.Y = otherParticle.m_Position.Y - particle.m_Position.Y;

			//magnitude of normal vector
			float magnitude = -1 * sqrt(normalVector.X * normalVector.X + normalVector.Y * normalVector.Y);

			GpuVector2D temporaryVelocity = -normalVector * particleRepulsionForce;

			particles[index].m_TemporaryVelocity.X = temporaryVelocity.X;
			particles[index].m_TemporaryVelocity.Y = temporaryVelocity.Y;

			//otherParticle->m_TemporaryVelocity = normalVector * particleRepulsionForce;
		}
	}
}

__device__ void updateParticleDensities(int index, Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	Range* lengths, int interactionMatrixRows, int interactionMatrixCols)
{

	Particle particle = particles[index];

	Vector2D point = particle.m_PredictedPosition;

	constexpr auto scalar = 1000;

	float density = 0.0f;
	const float mass = 1.0f;

	int row = point.Y / particleRadiusOfRepel;
	int col = point.X / particleRadiusOfRepel;

	for (int i = -1; i < 2; i++) {
		for (int j = -1; j < 2; j++) {
			if (row + i < 0 || row + i >= interactionMatrixRows || col + j < 0 || col + j >= interactionMatrixCols) {
				continue;
			}

			int lengthIndex = (row + i) * interactionMatrixCols + col + j;

			for (int otherParticleIndex = lengths[lengthIndex].start; otherParticleIndex < lengths[lengthIndex].end; otherParticleIndex++) {
				Particle otherParticle = particles[otherParticleIndex];
				float distance = sqrt(CudaMath::squared_distance(point, otherParticle.m_PredictedPosition));
				float influence = CudaMath::smoothingKernel(particleRadiusOfRepel, distance);
				density += mass * influence;
			}

		}
	}

	float volume = 3.1415f * pow(particleRadiusOfRepel, 2);

	if (density == 0) {
		//printf("densityyyyyyy1: %f, index: %d\n", density, index);
		density = mass * CudaMath::smoothingKernel(particleRadiusOfRepel, 0);
	}

	density = density / volume * scalar;

	/*if (density == 0) {
		printf("densityyyyyyy: %f, index: %d\n", density, index);
	}*/

	particles[index].m_Density = density;
}

__device__ GpuVector2D calculatePressureForce2(int index, Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	int particleRadius, Range* lengths, int interactionMatrixRows, int interactionMatrixCols)
{

	Particle particle = particles[index];

	Vector2D point = particle.m_PredictedPosition;

	int row = point.Y / particleRadiusOfRepel;
	int col = point.X / particleRadiusOfRepel;


	GpuVector2D pressureForce = GpuVector2D();
	const float mass = 1.0f;

	for (int i = -1; i < 2; i++) {
		for (int j = -1; j < 2; j++) {
			if (row + i < 0 || row + i >= interactionMatrixRows || col + j < 0 || col + j >= interactionMatrixCols) {
				continue;
			}

			int lengthIndex = (row + i) * interactionMatrixCols + col + j;

			for (int otherParticleIndex = lengths[lengthIndex].start; otherParticleIndex < lengths[lengthIndex].end; otherParticleIndex++) {
				Particle otherParticle = particles[otherParticleIndex];

				if (particle.m_ID == otherParticle.m_ID) {
					continue;
				}

				float distance = sqrt(CudaMath::squared_distance(particle.m_PredictedPosition, otherParticle.m_PredictedPosition));

				GpuVector2D dir = distance < particleRadius ? GpuVector2D::getRandomDirection() : (GpuVector2D(otherParticle.m_PredictedPosition) - GpuVector2D(particle.m_PredictedPosition)) / distance;

				float slope = CudaMath::smoothingKernelDerivative(particleRadiusOfRepel, distance);

				float density = otherParticle.m_Density;

				/*if (density == 0) {
					printf("density: %f, index: %d\n", density, index);
				}*/

				float sharedPressure = CudaMath::calculateSharedPressure(density, otherParticle.m_Density);

				pressureForce += -sharedPressure * dir * slope * mass / density;
			}
		}
	}
	/*printf("pressureForceX: %f , pressureForceY: %f, index: %d\n", pressureForce.X, pressureForce.Y, index);*/
	return pressureForce;
}

__device__ void updateParticleFutureVelocities(int index, Particle* particles, int praticlesSize,
	int particleRadiusOfRepel, int particleRadius, Range* lengths, size_t interactionMatrixRows,
	size_t interactionMatrixCols, double dt)
{

	Particle particle = particles[index];

	if (particle.m_Density == 0) {
		return;
	}

	GpuVector2D pressureForce = calculatePressureForce2(index, particles, praticlesSize, particleRadiusOfRepel, particleRadius,
		lengths, interactionMatrixRows, interactionMatrixCols);

	//pressureForce = GpuVector2D(300, 300);
	/*if (isnan(pressureForce.X) || isnan(pressureForce.Y)) {
		printf("pressureForceX: %f , pressureForceY: %f, index: %d\n", pressureForce.X, pressureForce.Y, index);
	}*/
	GpuVector2D pressureAcceleration = pressureForce / particle.m_Density;

	//Vector2D viscosityForce = calculateViscosityForce(particle);
	GpuVector2D viscosityForce = GpuVector2D();

	GpuVector2D futureVelocity = GpuVector2D(particle.m_Velocity) + pressureAcceleration * dt + viscosityForce * dt;

	//printf("index: %d, futureVelocity: %f %f \n", index, futureVelocity.X, futureVelocity.Y);

	particles[index].m_FutureVelocity.X = futureVelocity.X;
	particles[index].m_FutureVelocity.Y = futureVelocity.Y;


	particles[index].m_Velocity = particles[index].m_FutureVelocity;
	updateParticle(index, particles, dt);
	particles[index].m_PredictedPosition = particles[index].m_Position;
}

__device__ void updateCollisions(int index, Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	int particleRadius, float particleRepulsionForce, Range* lengths, int interactionMatrixRows,
	int interactionMatrixCols, Surface2D* obstacles, int obstaclesSize)
{


	Particle particle = particles[index];

	Vector2D point = particle.m_PredictedPosition;

	int row = point.Y / particleRadiusOfRepel;
	int col = point.X / particleRadiusOfRepel;

	for (int i = 0; i < obstaclesSize; i++) {

		Surface2D obstacle = obstacles[i];

		if (CudaMath::check_line_segment_circle_intersection(obstacle.Point1, obstacle.Point2,
			particle.m_Position, particleRadius)) {

			//particle->m_Velocity = reflectionVector * 0.1f;
			particles[index].m_Velocity.X = 0;
			particles[index].m_Velocity.Y = 0;

			/*particle.m_Velocity.Y = -particle.m_Velocity.Y;*/

			particles[index].m_Position = particle.m_LastSafePosition;

			break;
		}
	}


	for (int i = -1; i < 2; i++) {
		for (int j = -1; j < 2; j++) {
			if (row + i < 0 || row + i >= interactionMatrixRows || col + j < 0 || col + j >= interactionMatrixCols) {
				continue;
			}

			int lengthIndex = (row + i) * interactionMatrixCols + col + j;

			for (int otherParticleIndex = lengths[lengthIndex].start;
				otherParticleIndex < lengths[lengthIndex].end; otherParticleIndex++) {

				Particle otherParticle = particles[otherParticleIndex];

				if (particle.m_ID == otherParticle.m_ID) {
					continue;
				}

				if (CudaMath::squared_distance(particle.m_Position, otherParticle.m_Position) <=
					(particleRadius * particleRadius) * 4) {

					GpuVector2D normalVector{};
					normalVector.X = otherParticle.m_Position.X - particle.m_Position.X;
					normalVector.Y = otherParticle.m_Position.Y - particle.m_Position.Y;

					//magnitude of normal vector
					float magnitude = -1 * sqrt(normalVector.X * normalVector.X + normalVector.Y * normalVector.Y);

					GpuVector2D temporaryVelocity = -normalVector * particleRepulsionForce;

					particles[index].m_TemporaryVelocity.X = temporaryVelocity.X;
					particles[index].m_TemporaryVelocity.Y = temporaryVelocity.Y;

					//otherParticle->m_TemporaryVelocity = normalVector * particleRepulsionForce;
				}
			}
		}
	}
}

__global__ void specialUpdateKernel(Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	int particleRadius, float particleRepulsionForce, Range* lengths, int interactionMatrixRows,
	int interactionMatrixCols, Surface2D* obstacles, int obstaclesSize, double dt) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	//printf("important index: %d \n", index);

	if (index >= praticlesSize) {
		return;
	}

	//printf("index: %d \n", index);

	Particle particle = particles[index];

	GpuVector2D newPredictedPosition = GpuVector2D(particles[index].m_Position) +
		GpuVector2D(particles[index].m_Velocity) * dt * HOW_FAR_INTO_THE_FUTURE;

	//printf("index: %d, newPredictedPosition: %f %f \n", index, newPredictedPosition.X, newPredictedPosition.Y);

	particles[index].m_PredictedPosition.X = newPredictedPosition.X;
	particles[index].m_PredictedPosition.Y = newPredictedPosition.Y;

	// Synchronize all threads in the block
	atomicAdd(&counterPredictedPositionsDone, 1);

	while (counterPredictedPositionsDone < praticlesSize) {
		continue;
	}
	// calculate densities

	updateParticleDensities(index, particles, praticlesSize, particleRadiusOfRepel, lengths,
		interactionMatrixRows, interactionMatrixCols);

	//printf("index: %d, density: %f \n", index, particles[index].m_Density);

	// calculate densities

	atomicAdd(&counterDensitiesDone, 1);

	while (counterDensitiesDone < praticlesSize) {
		continue;
	}

	// calculate future velocities

	updateParticleFutureVelocities(index, particles, praticlesSize, particleRadiusOfRepel,
		particleRadius, lengths, interactionMatrixRows, interactionMatrixCols, dt);

	// calculate future velocities

	atomicAdd(&counterFutureVelocitiesDone, 1);

	while (counterFutureVelocitiesDone < praticlesSize) {
		continue;
	}

	// check collisions

	updateCollisions(index, particles, praticlesSize, particleRadiusOfRepel, particleRadius,
		particleRepulsionForce, lengths, interactionMatrixRows, interactionMatrixCols, obstacles, obstaclesSize);

	// check collisions


}

__global__ void resetGlobalCounter() {
	counterDensitiesDone = 0;
	counterPredictedPositionsDone = 0;
	counterFutureVelocitiesDone = 0;
}

void GpuAllocateInteractionMatrix(InteractionMatrixClass* interactionMatrix) {


	interactionMatrixSize = interactionMatrix->getMatrix().size() * interactionMatrix->getMatrix().at(0).size();

	maxParticlesInInteractionMatrixCell = 0;
	interactionMatrixRows = interactionMatrix->getMatrix().size();
	interactionMatrixCols = interactionMatrix->getMatrix().at(0).size();

	int* hostLengths = new int[interactionMatrixSize];

	for (int i = 0; i < interactionMatrixRows; i++) {
		for (int j = 0; j < interactionMatrixCols; j++) {
			hostLengths[i * interactionMatrixCols + j] = interactionMatrix->getMatrix().at(i).at(j).particles.size();
			if (interactionMatrix->getMatrix().at(i).at(j).particles.size() > maxParticlesInInteractionMatrixCell) {
				maxParticlesInInteractionMatrixCell = interactionMatrix->getMatrix().at(i).at(j).particles.size();
			}
		}
	}

	Particle* hostInteractionMatrixParticles = new Particle[interactionMatrixSize * maxParticlesInInteractionMatrixCell];

	for (int i = 0; i < interactionMatrixRows; i++) {
		for (int j = 0; j < interactionMatrixCols; j++) {
			int index = i * interactionMatrixCols + j;

			for (int k = 0; k < maxParticlesInInteractionMatrixCell; k++) {
				if (k < hostLengths[index]) {
					hostInteractionMatrixParticles[index * maxParticlesInInteractionMatrixCell + k] =
						*interactionMatrix->getMatrix().at(i).at(j).particles[k];
				}
				else {
					hostInteractionMatrixParticles[index * maxParticlesInInteractionMatrixCell + k] = Particle();
				}
			}
		}
	}

	// Allocate memory on GPU
	hipMalloc(&deviceLengths, interactionMatrixSize * sizeof(int));
	hipMemcpy(deviceLengths, hostLengths, interactionMatrixSize * sizeof(int), hipMemcpyHostToDevice);



	hipMalloc(&deviceInteractionMatrixParticles,
		interactionMatrixSize * maxParticlesInInteractionMatrixCell * sizeof(Particle));

	// Copy data from CPU to GPU
	hipMemcpy(deviceInteractionMatrixParticles, hostInteractionMatrixParticles,
		interactionMatrixSize * maxParticlesInInteractionMatrixCell * sizeof(Particle), hipMemcpyHostToDevice);

	// Free pointers
	delete[] hostLengths;
	delete[] hostInteractionMatrixParticles;
}

void GpuFreeInteractionMatrix() {
	hipFree(deviceInteractionMatrixParticles);
	hipFree(deviceLengths);
}

void GpuParallelUpdateParticleDensities(std::vector<Particle>& particles, int particleRadiusOfRepel) {

	// Allocate memory on GPU
	Particle* gpuParticles;

	hipMalloc(&gpuParticles, particles.size() * sizeof(Particle));

	// Copy data from CPU to GPU
	hipMemcpy(gpuParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

	int numThreads = particles.size();
	int maxThreadsPerBlock = 1024;

	int blockSize = maxThreadsPerBlock;
	int numBlocks = (numThreads + blockSize - 1) / blockSize;

	int blockSize2 = maxThreadsPerBlock;
	int numBlocks2 = particles.size();

	std::chrono::steady_clock::time_point time1 = std::chrono::steady_clock::now();

	// Launch CUDA kernel
	updateParticleDensitiesKernel << <numBlocks, blockSize >> > (gpuParticles, particles.size(), particleRadiusOfRepel,
		deviceInteractionMatrixParticles, deviceLengths, interactionMatrixRows, interactionMatrixCols,
		maxParticlesInInteractionMatrixCell);

	/*updateParticleDensitiesKernel2 << <numBlocks2, blockSize2 >> > (gpuParticles, particles.size(), particleRadiusOfRepel,
		deviceInteractionMatrixParticles, deviceLengths, interactionMatrixRows, interactionMatrixCols,
		maxParticlesInInteractionMatrixCell);*/

		// Wait for kernel to finish
	hipDeviceSynchronize();

	std::chrono::steady_clock::time_point time2 = std::chrono::steady_clock::now();
	double tick = std::chrono::duration_cast<std::chrono::microseconds>(time2 - time1).count();

	// Using std::unique_ptr to manage memory
	Particle* output = new Particle[particles.size()];

	hipMemcpy(output, gpuParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free output
	delete[] output;

	// Free GPU memory
	hipFree(gpuParticles);

}


void GpuParallelCalculateFutureVelocities(std::vector<Particle>& particles, int particleRadiusOfRepel,
	int particleRadius, double dt)
{

	// Allocate memory on GPU
	Particle* gpuParticles;

	hipMalloc(&gpuParticles, particles.size() * sizeof(Particle));

	// Copy data from CPU to GPU
	hipMemcpy(gpuParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

	int numThreads = particles.size();
	int maxThreadsPerBlock = 1024;

	int blockSize = maxThreadsPerBlock;
	int numBlocks = (numThreads + blockSize - 1) / blockSize;

	// Launch CUDA kernel
	calculateParticleFutureVelocitiesKernel << <numBlocks, blockSize >> > (gpuParticles, particles.size(), particleRadiusOfRepel,
		particleRadius, deviceInteractionMatrixParticles, deviceLengths, interactionMatrixRows, interactionMatrixCols,
		maxParticlesInInteractionMatrixCell, dt);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Using std::unique_ptr to manage memory
	Particle* output = new Particle[particles.size()];

	hipMemcpy(output, gpuParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free output
	delete[] output;

	// Free GPU memory
	hipFree(gpuParticles);

}

void GpuParallelCheckCollision(std::vector<Particle>& particles, int particleRadiusOfRepel,
	int particleRadius, float particleRepulsionForce, std::vector<Surface2D>& obstacles,
	double dt)
{

	// Allocate memory on GPU
	Particle* deviceParticles;
	Surface2D* deviceObstacles;

	hipMalloc(&deviceParticles, particles.size() * sizeof(Particle));
	hipMalloc(&deviceObstacles, obstacles.size() * sizeof(Surface2D));

	// Copy data from CPU to GPU
	hipMemcpy(deviceParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
	hipMemcpy(deviceObstacles, obstacles.data(), obstacles.size() * sizeof(Surface2D), hipMemcpyHostToDevice);

	int numThreads = particles.size();
	int maxThreadsPerBlock = 1024;

	int blockSize = maxThreadsPerBlock;
	int numBlocks = (numThreads + blockSize - 1) / blockSize;

	// Launch CUDA kernel
	checkCollisionsKernel << <numBlocks, blockSize >> > (deviceParticles, particles.size(), particleRadiusOfRepel,
		particleRadius, particleRepulsionForce, deviceInteractionMatrixParticles, deviceLengths, interactionMatrixRows,
		interactionMatrixCols, maxParticlesInInteractionMatrixCell, deviceObstacles, obstacles.size());

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Using std::unique_ptr to manage memory
	Particle* output = new Particle[particles.size()];

	hipMemcpy(output, deviceParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free output
	delete[] output;

	// Free GPU memory
	hipFree(deviceParticles);
	hipFree(deviceObstacles);

}

__device__ Range divideEtImpera(Particle* particles, int left, int right, int particlesSize,
	int particleRadiusOfRepel, int expectedPosition, int interactionMatrixCols) {

	do {
		if (left >= right) {
			return Range{ 0,0 };
		}

		int mid = left + (right - left) / 2;

		int row = particles[mid].m_Position.Y / particleRadiusOfRepel;
		int col = particles[mid].m_Position.X / particleRadiusOfRepel;

		int position = row * interactionMatrixCols + col;

		if (position == expectedPosition) {
			Range range{ 0,0 };

			for (int index = mid; index >= 0; index--) {
				int currentRow = particles[index].m_Position.Y / particleRadiusOfRepel;
				int currentCol = particles[index].m_Position.X / particleRadiusOfRepel;

				int currentPosition = currentRow * interactionMatrixCols + currentCol;

				if (currentPosition != expectedPosition) {
					range.start = index + 1;
					break;
				}
			}

			range.end = particlesSize;
			for (int index = mid; index < particlesSize; index++) {
				int currentRow = particles[index].m_Position.Y / particleRadiusOfRepel;
				int currentCol = particles[index].m_Position.X / particleRadiusOfRepel;

				int currentPosition = currentRow * interactionMatrixCols + currentCol;

				if (currentPosition != expectedPosition) {
					range.end = index;
					break;
				}
			}

			return range;
		}

		if (position < expectedPosition) {
			left = mid + 1;
		}
		else {
			right = mid - 1;
		}
	} while (true);
}

__global__ void setLengths(Particle* particles, int particlesSize, int particleRadiusOfRepel, Range* lengths, int interactionMatrixRows, int interactionMatrixCols) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= interactionMatrixRows * interactionMatrixCols) {
		return;
	}

	//printf("index: %d, start: %d, end: %d \n", index, lengths[index].start, lengths[index].end);
	lengths[index] = divideEtImpera(particles, 0, particlesSize - 1, particlesSize,
		particleRadiusOfRepel, index, interactionMatrixCols);

}

//GPU Kernel Implementation of Bitonic Sort
__global__ void bitonicSortGPU(Particle* arr, int j, int k, int particleRadiusOfRepel)
{
	unsigned int i, ij;

	i = threadIdx.x + blockDim.x * blockIdx.x;

	ij = i ^ j;

	if (ij > i)
	{
		int rowA = arr[i].m_Position.Y / particleRadiusOfRepel;
		int colA = arr[i].m_Position.X / particleRadiusOfRepel;

		int rowB = arr[ij].m_Position.Y / particleRadiusOfRepel;
		int colB = arr[ij].m_Position.X / particleRadiusOfRepel;

		bool lower;

		if (rowA == rowB) {
			lower = colA < colB;
		}
		else {
			lower = rowA < rowB;
		}

		if ((i & k) == 0)
		{
			if (!lower)
			{
				Particle temp = arr[i];
				arr[i] = arr[ij];
				arr[ij] = temp;
			}
		}
		else
		{
			if (lower)
			{
				Particle temp = arr[i];
				arr[i] = arr[ij];
				arr[ij] = temp;
			}
		}
	}
}

void GpuAllocate(std::vector<Particle>& particles, std::vector<Surface2D>& obstacles, int interactionMatrixSize) {

	hipError_t cudaStatus;

	// Allocate memory on GPU
	cudaStatus = hipMalloc(&deviceParticles, particles.size() * sizeof(Particle));
	cudaStatus = hipMalloc(&deviceObstacles, obstacles.size() * sizeof(Surface2D));

	// Copy data from CPU to GPU
	cudaStatus = hipMemcpy(deviceParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(deviceObstacles, obstacles.data(), obstacles.size() * sizeof(Surface2D), hipMemcpyHostToDevice);


	// Allocate memory on GPU
	cudaStatus = hipMalloc(&lengths, interactionMatrixSize * sizeof(Range));

	Range* hostLengths = new Range[interactionMatrixSize]{ Range{0,0} };
	cudaStatus = hipMemcpy(lengths, hostLengths, interactionMatrixSize * sizeof(Range), hipMemcpyHostToDevice);

	delete[] hostLengths;
}

void GpuFree() {
	// Free GPU memory
	hipFree(deviceParticles);
	hipFree(deviceObstacles);
	hipFree(lengths);
}

__global__ void demo(Particle* particles, int particlesSize) {
	printf("particlesSize: %d \n", particlesSize);
	/*for (int i = 0; i < particlesSize; i++) {
		printf("index: %d, position: %f %f \n", i, particles[i].m_Position.X, particles[i].m_Position.Y);
	}*/
}

void GpuUpdateParticles(std::vector<Particle>& particles, int particleRadiusOfRepel,
	int particleRadius, float particleRepulsionForce, std::vector<Surface2D>& obstacles,
	double dt, size_t interactionMatrixRows, size_t interactionMatrixCols,
	InteractionMatrixClass* interactionMatrix) {

	interactionMatrixSize = interactionMatrixRows * interactionMatrixCols;

	//hipMalloc(&deviceParticles, particles.size() * sizeof(Particle));
	//hipMemcpy(deviceParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

	//Set number of threads and blocks for kernel calls
	int threadsPerBlock = maxThreadsPerBlock;
	int blocksPerGrid = (particles.size() + threadsPerBlock - 1) / threadsPerBlock;
	int k, j;

	//demo << <1, 1 >> > (deviceParticles, particles.size());
	//hipDeviceSynchronize();

	// Bitonic Sort
	for (k = 2; k <= particles.size(); k <<= 1)
	{
		for (j = k >> 1; j > 0; j = j >> 1)
		{
			bitonicSortGPU << <blocksPerGrid, threadsPerBlock >> > (deviceParticles, j, k, particleRadiusOfRepel);
		}
	}
	hipDeviceSynchronize();

	//printf("\n\n\n 111111111111111111111111111111111 \n\n\n");

	//demo << <1, 1 >> > (deviceParticles, particles.size());
	//hipDeviceSynchronize();

	int blockSize = (interactionMatrixSize < maxThreadsPerBlock) ? interactionMatrixSize : maxThreadsPerBlock;
	int numBlocks = (interactionMatrixSize + blockSize - 1) / blockSize;

	// Launch CUDA kernel for setting lengths
	setLengths << < numBlocks, blockSize >> > (deviceParticles, particles.size(), particleRadiusOfRepel,
		lengths, interactionMatrixRows, interactionMatrixCols);

	// Wait for kernel to finish
	//hipDeviceSynchronize();

	//printf("\n\n\n 2222222222222222222222222222222222 \n\n\n");

	//demo << <1, 1 >> > (deviceParticles, particles.size());
	//hipDeviceSynchronize();

	resetGlobalCounter << <1, 1 >> > ();

	// Wait for kernel to finish
	hipDeviceSynchronize();

	blockSize = (particles.size() < maxThreadsPerBlock) ? particles.size() : maxThreadsPerBlock;
	numBlocks = (particles.size() + blockSize - 1) / blockSize;

	//printf("\n\n\n 33333333333333333333333333333333 \n\n\n");

	//demo << <1, 1 >> > (deviceParticles, particles.size());
	//hipDeviceSynchronize();

	// Launch CUDA kernel for updating particles
	specialUpdateKernel << <numBlocks, blockSize >> > (deviceParticles, particles.size(), particleRadiusOfRepel,
		particleRadius, particleRepulsionForce, lengths, interactionMatrixRows,
		interactionMatrixCols, deviceObstacles, obstacles.size(), dt);

	// Wait for kernel to finish
	hipDeviceSynchronize();


	Particle* output = new Particle[particles.size()];

	hipMemcpy(output, deviceParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free output
	delete[] output;
}
