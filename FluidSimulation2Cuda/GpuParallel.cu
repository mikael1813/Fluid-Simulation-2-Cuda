#include "hip/hip_runtime.h"
#include "GpuParallel.cuh"
#include "CudaMath.cuh"

#include <chrono>
#include "Phisics.hpp"

Particle* deviceInteractionMatrixParticles;

int* deviceLengths;
int interactionMatrixSize;

size_t maxParticlesInInteractionMatrixCell;
size_t interactionMatrixRows;
size_t interactionMatrixCols;

struct Range {
	int start;
	int end;
};

__device__ Range getParticlesInCell(Vector2D position, int particleRadiusOfRepel,
	int* lengths, size_t interactionMatrixRows, size_t interactionMatrixCols,
	size_t maxParticlesInInteractionMatrixCell)
{
	int row = position.Y / particleRadiusOfRepel;
	int col = position.X / particleRadiusOfRepel;

	if (row < 0 || row >= interactionMatrixRows || col < 0 || col >= interactionMatrixCols) {
		return;
	}

	int start = (row * interactionMatrixCols + col) * maxParticlesInInteractionMatrixCell;
	int end = start + lengths[row * interactionMatrixCols + col];

	return Range{ start, end };
}

__global__ void updateParticleDensitiesKernel(Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	Particle* interactionMatrixParticles, int* lengths, size_t interactionMatrixRows, size_t interactionMatrixCols,
	size_t maxParticlesInInteractionMatrixCell) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= praticlesSize) {
		return;
	}

	//printf("index: %d \n", index);

	Particle particle = particles[index];

	Vector2D point = particle.m_PredictedPosition;

	constexpr auto scalar = 1000;

	float density = 0.0f;
	const float mass = 1.0f;

	Range range = getParticlesInCell(point, particleRadiusOfRepel, lengths, interactionMatrixRows, interactionMatrixCols, maxParticlesInInteractionMatrixCell);

	//printf("index: %d, range.start: %d, range.end: %d \n", index, range.start, range.end);

	for (int i = range.start; i < range.end; i++) {
		Particle otherParticle = interactionMatrixParticles[i];
		float distance = sqrt(CudaMath::squared_distance(point, otherParticle.m_PredictedPosition));
		float influence = CudaMath::smoothingKernel(particleRadiusOfRepel, distance);
		density += mass * influence;
	}

	float volume = 3.1415f * pow(particleRadiusOfRepel, 2);

	density = density / volume * scalar;

	particles[index].m_Density = density;
}

__device__ GpuVector2D calculatePressureForce(Particle particle, int particleRadiusOfRepel, int particleRadius,
	Particle* interactionMatrixParticles, int* lengths, size_t interactionMatrixRows/*, size_t interactionMatrixCols,
	size_t maxParticlesInInteractionMatrixCell*/)
{

	GpuVector2D pressureForce = GpuVector2D();
	/*const float mass = 1.0f;

	Range range = getParticlesInCell(particle.m_PredictedPosition, particleRadiusOfRepel, lengths, interactionMatrixRows, interactionMatrixCols, maxParticlesInInteractionMatrixCell);

	for (int i = range.start; i < range.end; i++) {
		Particle otherParticle = interactionMatrixParticles[i];

		if (particle.m_ID == otherParticle.m_ID) {
			continue;
		}

		float distance = sqrt(CudaMath::squared_distance(particle.m_PredictedPosition, otherParticle.m_PredictedPosition));
		if (distance < particleRadius) {
			int tt = 0;
		}
		GpuVector2D dir = distance < particleRadius ? GpuVector2D::getRandomDirection() : (GpuVector2D(otherParticle.m_PredictedPosition) - GpuVector2D(particle.m_PredictedPosition)) / distance;

		float slope = CudaMath::smoothingKernelDerivative(particleRadiusOfRepel, distance);

		float density = otherParticle.m_Density;

		float sharedPressure = CudaMath::calculateSharedPressure(density, otherParticle.m_Density);

		pressureForce += -sharedPressure * dir * slope * mass / density;
	}*/

	return pressureForce;
}


__global__ void calculateParticleFutureVelocitiesKernel(Particle* particles, int praticlesSize, int particleRadiusOfRepel,
	int particleRadius, Particle* interactionMatrixParticles, int* lengths, size_t interactionMatrixRows,
	size_t interactionMatrixCols, size_t maxParticlesInInteractionMatrixCell, double dt)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= praticlesSize) {
		return;
	}

	//printf("index: %d \n", interactionMatrixCols);

	Particle particle = particles[index];

	if (particle.m_Density == 0) {
		return;
	}

	GpuVector2D pressureForce = calculatePressureForce(particle, particleRadiusOfRepel, particleRadius,
		interactionMatrixParticles, lengths, interactionMatrixRows/*, interactionMatrixCols,
		maxParticlesInInteractionMatrixCell*/);

	//pressureForce = GpuVector2D(300, 300);
	GpuVector2D pressureAcceleration = pressureForce / particle.m_Density;

	//Vector2D viscosityForce = calculateViscosityForce(particle);
	GpuVector2D viscosityForce = GpuVector2D();

	GpuVector2D futureVelocity = GpuVector2D(particle.m_Velocity) + pressureAcceleration * dt + viscosityForce * dt;

	//printf("index: %d, futureVelocity: %f %f \n", index, futureVelocity.X, futureVelocity.Y);

	particles[index].m_FutureVelocity.X = futureVelocity.X;
	particles[index].m_FutureVelocity.Y = futureVelocity.Y;
}

void GpuAllocateInteractionMatrix(InteractionMatrixClass* interactionMatrix) {
	//

	interactionMatrixSize = interactionMatrix->getMatrix().size() * interactionMatrix->getMatrix().at(0).size();

	maxParticlesInInteractionMatrixCell = 0;
	interactionMatrixRows = interactionMatrix->getMatrix().size();
	interactionMatrixCols = interactionMatrix->getMatrix().at(0).size();

	int* hostLengths = new int[interactionMatrixSize];

	for (int i = 0; i < interactionMatrixRows; i++) {
		for (int j = 0; j < interactionMatrixCols; j++) {
			hostLengths[i * interactionMatrixCols + j] = interactionMatrix->getMatrix().at(i).at(j).particles.size();
			if (interactionMatrix->getMatrix().at(i).at(j).particles.size() > maxParticlesInInteractionMatrixCell) {
				maxParticlesInInteractionMatrixCell = interactionMatrix->getMatrix().at(i).at(j).particles.size();
			}
		}
	}

	Particle* hostInteractionMatrixParticles = new Particle[interactionMatrixSize * maxParticlesInInteractionMatrixCell];

	for (int i = 0; i < interactionMatrixRows; i++) {
		for (int j = 0; j < interactionMatrixCols; j++) {
			int index = i * interactionMatrixCols + j;

			for (int k = 0; k < maxParticlesInInteractionMatrixCell; k++) {
				if (k < hostLengths[index]) {
					hostInteractionMatrixParticles[index * maxParticlesInInteractionMatrixCell + k] =
						*interactionMatrix->getMatrix().at(i).at(j).particles[k];
				}
				else {
					hostInteractionMatrixParticles[index * maxParticlesInInteractionMatrixCell + k] = Particle();
				}
			}
		}
	}

	// Allocate memory on GPU
	hipMalloc(&deviceLengths, interactionMatrixSize * sizeof(int));
	hipMemcpy(deviceLengths, hostLengths, interactionMatrixSize * sizeof(int), hipMemcpyHostToDevice);



	hipMalloc(&deviceInteractionMatrixParticles,
		interactionMatrixSize * maxParticlesInInteractionMatrixCell * sizeof(Particle));

	// Copy data from CPU to GPU
	hipMemcpy(deviceInteractionMatrixParticles, hostInteractionMatrixParticles,
		interactionMatrixSize * maxParticlesInInteractionMatrixCell * sizeof(Particle), hipMemcpyHostToDevice);

	// Free pointers
	delete[] hostLengths;
	delete[] hostInteractionMatrixParticles;
}

void GpuFreeInteractionMatrix() {
	hipFree(deviceInteractionMatrixParticles);
	hipFree(deviceLengths);
}

void GpuParallelUpdateParticleDensities(std::vector<Particle>& particles, int particleRadiusOfRepel) {

	// Allocate memory on GPU
	Particle* gpuParticles;

	hipMalloc(&gpuParticles, particles.size() * sizeof(Particle));

	// Copy data from CPU to GPU
	hipMemcpy(gpuParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

	int numThreads = particles.size();
	int maxThreadsPerBlock = 1024;

	int blockSize = maxThreadsPerBlock;
	int numBlocks = (numThreads + blockSize - 1) / blockSize;

	// Launch CUDA kernel
	updateParticleDensitiesKernel << <numBlocks, blockSize >> > (gpuParticles, particles.size(), particleRadiusOfRepel,
		deviceInteractionMatrixParticles, deviceLengths, interactionMatrixRows, interactionMatrixCols,
		maxParticlesInInteractionMatrixCell);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Using std::unique_ptr to manage memory
	Particle* output = new Particle[particles.size()];

	hipMemcpy(output, gpuParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free output
	delete[] output;

	// Free GPU memory
	hipFree(gpuParticles);

}


void GpuParallelCalculateFutureVelocities(std::vector<Particle>& particles, int particleRadiusOfRepel,
	int particleRadius, double dt)
{

	// Allocate memory on GPU
	Particle* gpuParticles;

	hipMalloc(&gpuParticles, particles.size() * sizeof(Particle));

	// Copy data from CPU to GPU
	hipMemcpy(gpuParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

	int numThreads = particles.size();
	int maxThreadsPerBlock = 1024;

	int blockSize = maxThreadsPerBlock;
	int numBlocks = (numThreads + blockSize - 1) / blockSize;

	// Launch CUDA kernel
	calculateParticleFutureVelocitiesKernel << <numBlocks, blockSize >> > (gpuParticles, particles.size(), particleRadiusOfRepel,
		particleRadius, deviceInteractionMatrixParticles, deviceLengths, interactionMatrixRows, interactionMatrixCols,
		maxParticlesInInteractionMatrixCell, dt);

	// Wait for kernel to finish
	hipDeviceSynchronize();

	// Using std::unique_ptr to manage memory
	Particle* output = new Particle[particles.size()];

	hipMemcpy(output, gpuParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free output
	delete[] output;

	// Free GPU memory
	hipFree(gpuParticles);

}

