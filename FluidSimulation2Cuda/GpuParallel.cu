#include "hip/hip_runtime.h"
#include "GpuParallel.cuh"

#include "Phisics.hpp"

__global__ void updateParticleDensitiesKernel(Particle* particles, int particleRadiusOfRepel) {

	int index = threadIdx.x;

	printf("index: %d", index);

	/*Particle particle = particles[index];

	Vector2D point = particle.m_PredictedPosition;

	particles[index].m_Density = 2.8;*/

	/*std::vector<Particle*> particlesInCell = interactionMatrix->getParticlesInCell(point, particleRadiusOfRepel);

	constexpr auto scalar = 1000;

	float density = 0.0f;
	const float mass = 1.0f;

	for (int i = 0; i < particlesInCell.size(); i++) {
		float distance = sqrt(Math::squared_distance(point, particle->m_PredictedPosition));
		float influence = Math::smoothingKernel(particleRadiusOfRepel, distance);
		density += mass * influence;
	}

	float volume = 3.1415f * pow(particleRadiusOfRepel, 2);

	density = density / volume * scalar;

	particle->m_Density = density;*/
}



void GpuParallelUpdateParticleDensities(std::vector<Particle>& particles, InteractionMatrixClass* interactionMatrix, int particleRadiusOfRepel) {


	// Allocate memory on GPU
	Particle* gpuParticles;
	hipMalloc(&gpuParticles, particles.size() * sizeof(Particle));

	// Copy data from CPU to GPU
	hipMemcpy(gpuParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

	// Launch CUDA kernel
	updateParticleDensitiesKernel << <1, particles.size() >> > (gpuParticles, particles.size());

	// Wait for kernel to finish
	hipDeviceSynchronize();

	Particle* output = new Particle[particles.size()];

	hipMemcpy(output, gpuParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free GPU memory
	hipFree(gpuParticles);

	//Particle* cudaParticles;

	//hipError_t cudaStatus;

	//InteractionMatrixClass* cudaInteractionMatrix;
	////int cudaParticleRadiusOfRepel;

	//cudaStatus = hipMalloc(&cudaParticles, particles.size() * sizeof(Particle));

	//cudaStatus = hipMemcpy(cudaParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

	//updateParticleDensitiesKernel << <1, particles.size() >> > (cudaParticles, particleRadiusOfRepel);

	//// Wait for the kernel to finish
	//hipDeviceSynchronize();

	//Particle* resultParticles = new Particle[particles.size()];
	//hipMemcpy(resultParticles, cudaParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	////cudaStatus = hipMemcpy(output, cudaParticles, particles.size() * sizeof(Particle*), hipMemcpyDeviceToHost);
	//auto y = resultParticles[9];
	//int x = 0;

	//// Cleanup resources
	//hipFree(cudaParticles);

	//x = 2;

}

// CUDA kernel function
__global__ void processParticlesKernel(Quo* particles, int numParticles) {
	int index = threadIdx.x;
	printf("index: %d\n", index);
	if (index < numParticles) {
		particles[index].density = index; // Process particle data here
	}
}

void processDataOnGPU(std::vector<Quo>& particles) {
	// Allocate memory on GPU
	Quo* gpuParticles;
	hipMalloc(&gpuParticles, particles.size() * sizeof(Quo));

	// Copy data from CPU to GPU
	hipMemcpy(gpuParticles, particles.data(), particles.size() * sizeof(Quo), hipMemcpyHostToDevice);

	// Launch CUDA kernel
	processParticlesKernel << <1, particles.size() >> > (gpuParticles, particles.size());

	// Wait for kernel to finish
	hipDeviceSynchronize();

	Quo* output = new Quo[particles.size()];

	hipMemcpy(output, gpuParticles, particles.size() * sizeof(Quo), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free GPU memory
	hipFree(gpuParticles);
}