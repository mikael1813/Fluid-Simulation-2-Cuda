#include "hip/hip_runtime.h"
#include "GpuParallel.cuh"

#include "Phisics.hpp"

__global__ void updateParticleDensitiesKernel(Particle* particles, int particleRadiusOfRepel) {

	int index = threadIdx.x;

	printf("index: %d", index);

	/*Particle particle = particles[index];

	Vector2D point = particle.m_PredictedPosition;

	particles[index].m_Density = 2.8;*/

	/*std::vector<Particle*> particlesInCell = interactionMatrix->getParticlesInCell(point, particleRadiusOfRepel);

	constexpr auto scalar = 1000;

	float density = 0.0f;
	const float mass = 1.0f;

	for (int i = 0; i < particlesInCell.size(); i++) {
		float distance = sqrt(Math::squared_distance(point, particle->m_PredictedPosition));
		float influence = Math::smoothingKernel(particleRadiusOfRepel, distance);
		density += mass * influence;
	}

	float volume = 3.1415f * pow(particleRadiusOfRepel, 2);

	density = density / volume * scalar;

	particle->m_Density = density;*/
}



void GpuParallelUpdateParticleDensities(std::vector<Particle>& particles, InteractionMatrixClass* interactionMatrix, int particleRadiusOfRepel) {


	Particle* cudaParticles;

	hipError_t cudaStatus;

	InteractionMatrixClass* cudaInteractionMatrix;
	//int cudaParticleRadiusOfRepel;

	cudaStatus = hipMalloc(&cudaParticles, particles.size() * sizeof(Particle));
	//cudaStatus = hipMalloc(&cudaInteractionMatrix, sizeof(InteractionMatrixClass));
	//cudaStatus = hipMalloc(&cudaParticleRadiusOfRepel, sizeof(int));

	cudaStatus = hipMemcpy(cudaParticles, particles.data(), particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(cudaInteractionMatrix, interactionMatrix, sizeof(InteractionMatrixClass), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(cudaParticleRadiusOfRepel, &particleRadiusOfRepel, sizeof(int), hipMemcpyHostToDevice);

	updateParticleDensitiesKernel << <1, particles.size() >> > (cudaParticles, particleRadiusOfRepel);

	// Wait for the kernel to finish
	hipDeviceSynchronize();

	Particle* resultParticles = new Particle[particles.size()];
	hipMemcpy(resultParticles, cudaParticles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

	//cudaStatus = hipMemcpy(output, cudaParticles, particles.size() * sizeof(Particle*), hipMemcpyDeviceToHost);
	auto y = resultParticles[9];
	int x = 0;

	// Cleanup resources
	hipFree(cudaParticles);

	x = 2;

}

// CUDA kernel function
__global__ void processParticlesKernel(Quo* particles, int numParticles) {
	int index = threadIdx.x;
	printf("index: %d\n", index);
	if (index < numParticles) {
		particles[index].density = index; // Process particle data here
	}
}

void processDataOnGPU(std::vector<Quo>& particles) {
	// Allocate memory on GPU
	Quo* gpuParticles;
	hipMalloc(&gpuParticles, particles.size() * sizeof(Quo));

	// Copy data from CPU to GPU
	hipMemcpy(gpuParticles, particles.data(), particles.size() * sizeof(Quo), hipMemcpyHostToDevice);

	// Launch CUDA kernel
	processParticlesKernel << <1, particles.size() >> > (gpuParticles, particles.size());

	// Wait for kernel to finish
	hipDeviceSynchronize();

	Quo* output = new Quo[particles.size()];

	hipMemcpy(output, gpuParticles, particles.size() * sizeof(Quo), hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); i++) {
		particles[i] = output[i];
	}

	// Free GPU memory
	hipFree(gpuParticles);
}